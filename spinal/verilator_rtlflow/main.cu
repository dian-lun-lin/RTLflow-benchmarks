#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <chrono>
#include "./obj_dir_rtlflow/VExampleTop.h"
//#include "./verilator/VExampleTop.h"

//#ifdef TRACE_VCD
//#include "rf_verilated_vcd_c.h"
//#endif

#include "../../../verilator_rtlflow/include/rf_verilated.h"
//#include "verilated.h"
//#include "../../../verilator/include/verilated.h"

RF::RTLflow rtlflow(NUM_TESTBENCHES);
RF::RTLflow& RF::VExampleTop::_rtlflow = rtlflow;
//void check(RF::RTLflow& rtlflow, VExampleTop__Syms* vlSymsp);

int main(int argc, char **argv) 
{
    RF::Verilated::commandArgs(argc, argv);

    int nr_cycles = 1000000;
    if (argc ==2)
        nr_cycles = atoi(argv[1]);
    //std::chrono::time_point<std::chrono::steady_clock> beg_sim;
    //std::chrono::time_point<std::chrono::steady_clock> end_sim;
    //double sim_time{0};

    std::cout << "Number of  cycles: " << nr_cycles << "\n";
    std::cout << "Number of testbenches: " << NUM_TESTBENCHES << "\n";

    //beg_sim = std::chrono::steady_clock::now();

    RF::VExampleTop *tb = new RF::VExampleTop;
    //VExampleTop *tb_cpu = new VExampleTop;
    bool prev_led_red;
    bool prev_led_green;
    bool prev_led_blue;
//#if defined(TRACE_VCD)
    //VerilatedVcdC *trace;
//#endif


//#if defined(TRACE_VCD)
    //Verilated::traceEverOn(true);
//#endif

//#if defined(TRACE_VCD)
    //trace = new VerilatedVcdC;
    //tb->trace(trace, 99);
    //trace->open("waves.vcd");
//#endif

    tb->eval();
    //tb_cpu->eval();
    //check(rtlflow, tb_cpu->__VlSymsp);

    //rtlflow._isignals[tb->button] = 1;
    //double set_time{0};
    //std::chrono::time_point<std::chrono::steady_clock> beg_set;
    //std::chrono::time_point<std::chrono::steady_clock> end_set;

    for(int i=0;i<nr_cycles;++i) {

      //beg_set = std::chrono::steady_clock::now();
      for(int t = 0; t < NUM_TESTBENCHES; ++t) {
        *(rtlflow.get(tb->osc_clk_in, t)) = 0;
        //tb_cpu->osc_clk_in = 0;
        //rtlflow._csignals[tb->osc_clk + t] = 0;
      }
      //end_set = std::chrono::steady_clock::now();
      //set_time += std::chrono::duration_cast<std::chrono::microseconds>(end_set - beg_set).count();

      tb->eval();
    //tb_cpu->eval();


      //beg_set = std::chrono::steady_clock::now();
      for(int t = 0; t < NUM_TESTBENCHES; ++t) {
        //rtlflow._csignals[tb->osc_clk + t] = 1;
        *(rtlflow.get(tb->osc_clk_in, t)) = 1;
        //tb_cpu->osc_clk_in = 1;
      }
      //end_set = std::chrono::steady_clock::now();
      //set_time += std::chrono::duration_cast<std::chrono::microseconds>(end_set - beg_set).count();

      //beg_sim = std::chrono::steady_clock::now();
      tb->eval();
    //tb_cpu->eval();
      //end_sim = std::chrono::steady_clock::now();
      //sim_time += std::chrono::duration_cast<std::chrono::microseconds>(end_sim - beg_sim).count();
      //std::cout << "clock: " << (int)rtlflow._csignals[tb->osc_clk_in] << "\n";

//#if defined(TRACE_VCD) || defined(TRACE_FST)
        //trace->dump(i*2);
//#endif
        //rtlflow._csignals[tb->osc_clk_in] = 1;

//#if defined(TRACE_VCD) || defined(TRACE_FST)
        //trace->dump(i*2);
//#endif

        //check(rtlflow, tb_cpu->__VlSymsp);

        bool cur_led_red    = *(rtlflow.get(tb->led_red, 0));
        bool cur_led_blue    = *(rtlflow.get(tb->led_blue, 0));
        bool cur_led_green    = *(rtlflow.get(tb->led_green, 0));
        //bool cur_led_blue    = rtlflow._csignals[tb->led_blue];
        //bool cur_led_green    = rtlflow._csignals[tb->led_green];

      //if(i > 0) {
      //for(int t = 0; t < 4095; ++t) {
        ////std::cerr << t << ", ";
        //assert(rtlflow._csignals[tb->led_red + t] == rtlflow._csignals[tb->led_red + t + 1]);
        //assert(rtlflow._csignals[tb->led_blue + t] == rtlflow._csignals[tb->led_blue + t + 1]);
        //assert(rtlflow._csignals[tb->led_green + t] == rtlflow._csignals[tb->led_green + t + 1]);
      //}
      //}

        //if (cur_led_red != prev_led_red){
            //std::cout << "led_red: " << cur_led_red << "\n";
        //}

        //if (cur_led_green != prev_led_green){
            //std::cout << "led_green: " << cur_led_green << "\n";
        //}

        //if (cur_led_blue != prev_led_blue){
            //std::cout << "led_blue: " << cur_led_blue << "\n";
        //}

        prev_led_red    = cur_led_red;
        prev_led_green  = cur_led_green;
        prev_led_blue   = cur_led_blue;

    }

    //end_sim = std::chrono::steady_clock::now();
    //sim_time = std::chrono::duration_cast<std::chrono::microseconds>(end_sim - beg_sim).count();
       
    //std::cout << "sim time: " << sim_time << "\n";
        
    exit(EXIT_SUCCESS);
}
