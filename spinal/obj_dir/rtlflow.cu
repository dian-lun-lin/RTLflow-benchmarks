#include "hip/hip_runtime.h"

#include <taskflow.hpp>

#include "rtlflow.h"


#include "VExampleTop.h"

#include <assert.h>

inline
hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        using namespace std::literals::string_literals;
        throw std::runtime_error("CUDA Runtime Error: "s + hipGetErrorString(result));
    }
    return result;
}

__global__ void _eval_settle(VExampleTop__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals);

// idx: index of testbenches
CData* RTLflow::get(CDataLoc cdl, size_t idx) {
    return _csignals + idx * cdl.size + cdl.memloc;
}
SData* RTLflow::get(SDataLoc sdl, size_t idx) {
    return _ssignals + idx * sdl.size + sdl.memloc;
}
QData* RTLflow::get(QDataLoc qdl, size_t idx) {
    return _qsignals + idx * qdl.size + qdl.memloc;
}
IData* RTLflow::get(IDataLoc idl, size_t idx) {
    return _isignals + idx * idl.size + idl.memloc;
}
RTLflow::RTLflow(size_t num_testbenches):num_testbenches{num_testbenches} {
    checkCuda(hipMallocManaged(&_csignals, num_testbenches * cuda_cmem_size * sizeof(CData)));
    checkCuda(hipMallocManaged(&_ssignals, num_testbenches * cuda_smem_size * sizeof(SData)));
    checkCuda(hipMallocManaged(&_qsignals, num_testbenches * cuda_qmem_size * sizeof(QData)));
    checkCuda(hipMallocManaged(&_isignals, num_testbenches * cuda_imem_size * sizeof(IData)));
    checkCuda(hipMallocManaged(&change, num_testbenches * sizeof(IData)));
    checkCuda(hipMemset(change, 1, num_testbenches * sizeof(IData)));
}
RTLflow::~RTLflow() {
    checkCuda(hipFree(_csignals));
    checkCuda(hipFree(_ssignals));
    checkCuda(hipFree(_qsignals));
    checkCuda(hipFree(_isignals));
    checkCuda(hipFree(change));
}
void RTLflow::run() { _executor.run(_taskflow).wait(); }
void RTLflow::initialize(VExampleTop__Syms* VlSymsp) {
    size_t num_threads = (num_testbenches < 1024) ? num_testbenches : 1024;
    size_t num_blocks = (num_threads < 1024) ? 1 : num_testbenches / num_threads;
    auto change_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, _change_request, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto last_assign_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, _last_assign, _csignals, _ssignals, _isignals, _qsignals);
    auto reduce_cut = _cudaflow.reduce(change, change + num_testbenches, change, [] __device__ (IData a, IData b){ return a | b; });
    last_assign_cut.precede(change_cut);
    
    change_cut.precede(reduce_cut);
    
    auto id_1_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__1, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_3_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__3, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_4_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__4, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_5_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__5, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_6_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__6, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_9_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__9, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_10_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__10, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_12_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__12, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_8_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__8, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_13_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__13, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_2_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__2, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_7_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__7, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_11_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__11, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_14_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__14, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_15_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__15, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_16_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__16, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_17_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__17, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_18_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__18, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_19_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__19, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    auto id_20_cut = _cudaflow.kernel(dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0, __Vmtask__20, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
    id_1_cut.precede(id_3_cut);
    id_1_cut.precede(id_4_cut);
    id_1_cut.precede(id_5_cut);
    id_1_cut.precede(id_6_cut);
    id_1_cut.precede(id_9_cut);
    id_1_cut.precede(id_10_cut);
    id_3_cut.precede(id_13_cut);
    id_4_cut.precede(id_13_cut);
    id_5_cut.precede(id_12_cut);
    id_5_cut.precede(id_8_cut);
    id_6_cut.precede(id_13_cut);
    id_9_cut.precede(id_12_cut);
    id_9_cut.precede(id_8_cut);
    id_10_cut.precede(id_13_cut);
    id_12_cut.precede(id_13_cut);
    id_8_cut.precede(id_13_cut);
    id_13_cut.precede(id_2_cut);
    id_13_cut.precede(id_7_cut);
    id_13_cut.precede(id_11_cut);
    id_2_cut.precede(id_14_cut);
    id_2_cut.precede(id_17_cut);
    id_7_cut.precede(id_14_cut);
    id_11_cut.precede(id_14_cut);
    id_11_cut.precede(id_17_cut);
    id_14_cut.precede(id_15_cut);
    id_14_cut.precede(id_16_cut);
    id_14_cut.precede(id_18_cut);
    id_14_cut.precede(id_19_cut);
    id_15_cut.precede(last_assign_cut);
    id_16_cut.precede(id_20_cut);
    id_17_cut.precede(id_20_cut);
    id_18_cut.precede(last_assign_cut);
    id_19_cut.precede(id_20_cut);
    id_20_cut.precede(last_assign_cut);
    auto start_t = _taskflow.emplace([=](){
            if(VL_UNLIKELY(!init)) {
                VExampleTop::_eval_initial(VlSymsp, _csignals, _ssignals, _isignals, _qsignals);
                int device;
                checkCuda(hipGetDevice(&device));
                checkCuda(hipMemPrefetchAsync(_csignals, num_testbenches * cuda_cmem_size * sizeof(CData), device));
                checkCuda(hipMemPrefetchAsync(_ssignals, num_testbenches * cuda_smem_size * sizeof(SData), device));
                checkCuda(hipMemPrefetchAsync(_isignals, num_testbenches * cuda_imem_size * sizeof(IData), device));
                checkCuda(hipMemPrefetchAsync(_qsignals, num_testbenches * cuda_qmem_size * sizeof(QData), device));
                checkCuda(hipMemPrefetchAsync(change, num_testbenches * sizeof(IData), device));
                init = true;
                return 0;
            }
            else {
                return 1;
            }
    });
    
    auto init_detect_t = _taskflow.emplace([=](){
            if(++loop > 100) {
                _change_request<<<dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0>>>(VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
                checkCuda(hipDeviceSynchronize());
                VL_FATAL_MT("add.v", 2, "",
                    "Verilated model didn't converge"
                    "- See https://verilator.org/warn/DIDNOTCONVERGE");
            }
            return (bool)change[0];
    });
    auto init_sim_t = _taskflow.emplace([=](){
            _eval_settle<<<dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0>>>(VlSymsp, _csignals, _ssignals, _isignals, _qsignals);
            checkCuda(hipDeviceSynchronize());
            _cudaflow.offload();
    });
    auto sim_t = _taskflow.emplace([=](){
            _cudaflow.offload();
    });
    auto end_t = _taskflow.emplace([=](){
            loop = 0;
            checkCuda(hipMemset(change, 1, sizeof(IData) * num_testbenches));
    });
    
    auto detect_t = _taskflow.emplace([=](){
            if(++loop > 100) {
                _change_request<<<dim3(num_blocks, 1, 1), dim3(num_threads, 1, 1), 0>>>(VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
                checkCuda(hipDeviceSynchronize());
                VL_FATAL_MT("add.v", 2, "",
                    "Verilated model didn't converge"
                    "- See https://verilator.org/warn/DIDNOTCONVERGE");
            }
            return (bool)change[0];
    });
    start_t.precede(init_sim_t, sim_t);
    init_sim_t.precede(init_detect_t);
    init_detect_t.precede(end_t, init_sim_t);
    
    sim_t.precede(detect_t);
    detect_t.precede(end_t, sim_t);
}
