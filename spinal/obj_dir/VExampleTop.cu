#include "hip/hip_runtime.h"
// Verilated -*- C++ -*-
// DESCRIPTION: Verilator output: Design implementation internals
// See VExampleTop.h for the primary calling header

#include "VExampleTop.h"
#include "VExampleTop__Syms.h"

#include "verilated_dpi.h"
#include "rtlflow.h"

//==========

VerilatedContext* VExampleTop::contextp() {
    return __VlSymsp->_vm_contextp__;
}

void VExampleTop::eval_step() {
    _rtlflow.run();
}

__global__
void _last_assign(CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
    VL_DEBUG_IF(VL_DBG_MSGF("+    VExampleTop::_last_assign\n"); );
    // Body
    // Final
    _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583] 
        = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576];
}

__global__
void _change_request(VExampleTop__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    VL_DEBUG_IF(VL_DBG_MSGF("+    VExampleTop::_change_request\n"); );
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    // Body
    IData __req = false;
    __req |= vlTOPp->_change_request_1(_csignals, _ssignals, _isignals, _qsignals);
    change[blockDim.x * blockIdx.x + threadIdx.x] = __req;
}

__device__
IData VExampleTop::_change_request_1(CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
    VL_DEBUG_IF(VL_DBG_MSGF("+    VExampleTop::_change_request_1\n"); );
    // Body
    // Change detection
    IData __req = false;  // Logically a bool
    return __req;
}

#ifdef VL_DEBUG
void VExampleTop::_eval_debug_assertions() {
    VL_DEBUG_IF(VL_DBG_MSGF("+    VExampleTop::_eval_debug_assertions\n"); );
    for(size_t i = 0; i < NUM_TESTBENCHES; ++i) {
        // Body
        if (VL_UNLIKELY((_csignals[i + NUM_TESTBENCHES * 576] 
                         & 0xfeU))) {
            Verilated::overWidthError("osc_clk_in");}
        if (VL_UNLIKELY((_csignals[i + NUM_TESTBENCHES * 577] 
                         & 0xfeU))) {
            Verilated::overWidthError("button");}
        if (VL_UNLIKELY((_csignals[i + NUM_TESTBENCHES * 582] 
                         & 0xfeU))) {
            Verilated::overWidthError("uart_rxd");}
    }
}
#endif  // VL_DEBUG

__global__
void __Vmtask__1(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__38(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__3(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__39(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__4(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__40(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__5(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__41(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__6(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__42(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__9(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__43(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__10(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu._sequent__TOP__ExampleTop__cpu_u_cpu__34(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__12(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__44(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__8(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__45(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__13(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__46(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__2(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__47(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__7(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__48(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__11(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__49(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__14(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__50(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__15(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu._sequent__TOP__ExampleTop__cpu_u_cpu__35(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__16(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__51(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__17(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__52(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__18(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._sequent__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__53(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__19(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop._sequent__TOP__ExampleTop__4(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}

__global__
void __Vmtask__20(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
    if(!change[blockDim.x * blockIdx.x + threadIdx.x]) return;
    VExampleTop__Syms* __restrict vlSymsp = (VExampleTop__Syms*)symtab;
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 576]) 
         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 583])))) {
        vlSymsp->TOP__ExampleTop__cpu_u_cpu._sequent__TOP__ExampleTop__cpu_u_cpu__36(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
}
