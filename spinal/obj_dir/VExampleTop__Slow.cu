#include "hip/hip_runtime.h"
// Verilated -*- C++ -*-
// DESCRIPTION: Verilator output: Design implementation internals
// See VExampleTop.h for the primary calling header

#include "VExampleTop.h"
#include "VExampleTop__Syms.h"

#include "verilated_dpi.h"
#include "rtlflow.h"

//==========

VExampleTop::VExampleTop(VerilatedContext* _vcontextp__, const char* _vcname__)
    : VerilatedModule{_vcname__}
 {
    VExampleTop__Syms* __restrict vlSymsp = __VlSymsp = new VExampleTop__Syms(_vcontextp__, this, _rtlflow._csignals, _rtlflow._ssignals, _rtlflow._isignals, _rtlflow._qsignals, name());
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    VL_CELL(ExampleTop, VExampleTop_ExampleTop);
    // Reset internal values
    _rtlflow.initialize(vlSymsp);
    

    // Reset structure values
    _ctor_var_reset(this, _rtlflow._csignals, _rtlflow._ssignals, _rtlflow._isignals, _rtlflow._qsignals);
}

void VExampleTop::__Vconfigure(VExampleTop__Syms* vlSymsp, bool first) {
    if (false && first) {}  // Prevent unused
    this->__VlSymsp = vlSymsp;
    if (false && this->__VlSymsp) {}  // Prevent unused
    vlSymsp->_vm_contextp__->timeunit(-12);
    vlSymsp->_vm_contextp__->timeprecision(-12);
}

VExampleTop::~VExampleTop() {
    VL_DO_CLEAR(delete __VlSymsp, __VlSymsp = nullptr);
}

__device__
void VExampleTop::_settle__TOP__1(VExampleTop__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
    VL_DEBUG_IF(VL_DBG_MSGF("+    VExampleTop::_settle__TOP__1\n"); );
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    // Body
    _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 581] 
        = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 8994];
}

__device__
void VExampleTop::_settle__TOP__2(VExampleTop__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
    VL_DEBUG_IF(VL_DBG_MSGF("+    VExampleTop::_settle__TOP__2\n"); );
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    // Body
    _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 580] 
        = (1U & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 8995]) 
                 >> 2U));
    _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 579] 
        = (1U & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 8995]) 
                 >> 1U));
    _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 578] 
        = (1U & (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + NUM_TESTBENCHES * 8995]));
}

void VExampleTop::_eval_initial(VExampleTop__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
    VL_DEBUG_IF(VL_DBG_MSGF("+    VExampleTop::_eval_initial\n"); );
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    // Body
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._initial__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__1(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._initial__TOP__ExampleTop__cpu_u_cpu__u_cpu__1(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop._initial__TOP__ExampleTop__1(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._initial__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__2(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    for(size_t i = 0; i < NUM_TESTBENCHES; ++i) {
        // Final
        _csignals[i + NUM_TESTBENCHES * 583] = _csignals[i + NUM_TESTBENCHES * 576];
    }
}

void VExampleTop::final() {
    VL_DEBUG_IF(VL_DBG_MSGF("+    VExampleTop::final\n"); );
    // Variables
    for(size_t i = 0; i < NUM_TESTBENCHES; ++i) {
        VExampleTop__Syms* __restrict vlSymsp = this->__VlSymsp;
        VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    }
}

__global__
void _eval_settle(VExampleTop__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
    VL_DEBUG_IF(VL_DBG_MSGF("+    VExampleTop::_eval_settle\n"); );
    VExampleTop* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
    // Body
    vlTOPp->_settle__TOP__1(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop._settle__TOP__ExampleTop__2(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__2(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__3(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__3(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__1(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__4(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__2(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__5(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__3(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__6(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__4(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__4(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__5(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__7(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__6(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__5(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__8(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__6(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__9(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__7(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__10(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__8(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__11(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__9(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__12(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlTOPp->_settle__TOP__2(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__13(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__10(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__14(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__11(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__15(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__12(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__16(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__13(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__17(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__14(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__18(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__15(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__19(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__16(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__20(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__17(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__21(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__18(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__22(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__19(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__23(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__20(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__24(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__21(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__25(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__22(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__26(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__23(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__27(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__7(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__24(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__28(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__25(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__29(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__26(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__30(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__8(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__27(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__9(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__31(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__28(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__32(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__29(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__10(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop._settle__TOP__ExampleTop__3(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__11(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__30(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__33(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__31(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__12(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__32(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__13(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__34(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__33(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__14(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__35(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__15(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__36(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__16(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__cpu__37(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    vlSymsp->TOP__ExampleTop__cpu_u_cpu__u_cpu._settle__TOP__ExampleTop__cpu_u_cpu__u_cpu__17(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
}

void VExampleTop::_ctor_var_reset(VExampleTop* self, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
    VL_DEBUG_IF(VL_DBG_MSGF("+    VExampleTop::_ctor_var_reset\n"); );
    for(size_t i = 0; i < NUM_TESTBENCHES; ++i) {
        // Body
        if (false && self) {}  // Prevent unused
        _csignals[i + 576 * NUM_TESTBENCHES] = VL_RAND_RESET_I(1);
        _csignals[i + 577 * NUM_TESTBENCHES] = VL_RAND_RESET_I(1);
        _csignals[i + 578 * NUM_TESTBENCHES] = VL_RAND_RESET_I(1);
        _csignals[i + 579 * NUM_TESTBENCHES] = VL_RAND_RESET_I(1);
        _csignals[i + 580 * NUM_TESTBENCHES] = VL_RAND_RESET_I(1);
        _csignals[i + 581 * NUM_TESTBENCHES] = VL_RAND_RESET_I(1);
        _csignals[i + 582 * NUM_TESTBENCHES] = VL_RAND_RESET_I(1);
    }
}
