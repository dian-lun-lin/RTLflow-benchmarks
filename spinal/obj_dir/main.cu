#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <chrono>
#include "VExampleTop.h"

#ifdef TRACE_VCD
#include "verilated_vcd_c.h"
#endif

#include "verilated.h"

RTLflow rtlflow(1);
RTLflow& VExampleTop::_rtlflow = rtlflow;

int main(int argc, char **argv) 
{
    Verilated::commandArgs(argc, argv);

    int nr_cycles = 1000000;
    if (argc ==2)
        nr_cycles = atoi(argv[1]);

    VExampleTop *tb = new VExampleTop;
    bool prev_led_red;
    bool prev_led_green;
    bool prev_led_blue;
//#if defined(TRACE_VCD)
    //VerilatedVcdC *trace;
//#endif


//#if defined(TRACE_VCD)
    //Verilated::traceEverOn(true);
//#endif

//#if defined(TRACE_VCD)
    //trace = new VerilatedVcdC;
    //tb->trace(trace, 99);
    //trace->open("waves.vcd");
//#endif

    //tb->eval();
    tb->eval();

    //rtlflow._isignals[tb->button] = 1;
    //double set_time{0};
    //double sim_time{0};
    //std::chrono::time_point<std::chrono::steady_clock> beg_set;
    //std::chrono::time_point<std::chrono::steady_clock> end_set;
    //std::chrono::time_point<std::chrono::steady_clock> beg_sim;
    //std::chrono::time_point<std::chrono::steady_clock> end_sim;

    for(int i=0;i<nr_cycles;++i) {

      //beg_set = std::chrono::steady_clock::now();
      for(int t = 0; t < 1; ++t) {
        rtlflow._csignals[tb->osc_clk_in.memloc + t] = 0;
      }
      //end_set = std::chrono::steady_clock::now();
      //set_time += std::chrono::duration_cast<std::chrono::microseconds>(end_set - beg_set).count();

      //beg_sim = std::chrono::steady_clock::now();
      tb->eval();
      //end_sim = std::chrono::steady_clock::now();
      //sim_time += std::chrono::duration_cast<std::chrono::microseconds>(end_sim - beg_sim).count();


      //beg_set = std::chrono::steady_clock::now();
      for(int t = 0; t < 1; ++t) {
        rtlflow._csignals[tb->osc_clk_in.memloc + t] = 1;
      }
      //end_set = std::chrono::steady_clock::now();
      //set_time += std::chrono::duration_cast<std::chrono::microseconds>(end_set - beg_set).count();

      //beg_sim = std::chrono::steady_clock::now();
      tb->eval();
      //end_sim = std::chrono::steady_clock::now();
      //sim_time += std::chrono::duration_cast<std::chrono::microseconds>(end_sim - beg_sim).count();
      //std::cout << "clock: " << (int)rtlflow._csignals[tb->osc_clk_in] << "\n";

//#if defined(TRACE_VCD) || defined(TRACE_FST)
        //trace->dump(i*2);
//#endif
        //rtlflow._csignals[tb->osc_clk_in] = 1;

//#if defined(TRACE_VCD) || defined(TRACE_FST)
        //trace->dump(i*2);
//#endif


        bool cur_led_red    = rtlflow._csignals[tb->led_red.memloc];
        bool cur_led_blue    = rtlflow._csignals[tb->led_blue.memloc];
        bool cur_led_green    = rtlflow._csignals[tb->led_green.memloc];

      //if(i > 0) {
      //for(int t = 0; t < 4095; ++t) {
        ////std::cerr << t << ", ";
        //assert(rtlflow._csignals[tb->led_red + t] == rtlflow._csignals[tb->led_red + t + 1]);
        //assert(rtlflow._csignals[tb->led_blue + t] == rtlflow._csignals[tb->led_blue + t + 1]);
        //assert(rtlflow._csignals[tb->led_green + t] == rtlflow._csignals[tb->led_green + t + 1]);
      //}
      //}

        //if (cur_led_red != prev_led_red){
            //std::cout << "led_red: " << cur_led_red << "\n";
        //}

        //if (cur_led_green != prev_led_green){
            //std::cout << "led_green: " << cur_led_green << "\n";
        //}

        //if (cur_led_blue != prev_led_blue){
            //std::cout << "led_blue: " << cur_led_blue << "\n";
        //}

        prev_led_red    = cur_led_red;
        prev_led_green  = cur_led_green;
        prev_led_blue   = cur_led_blue;

    }
       

    //std::cout << "set time: " << set_time << "\n";
    //std::cout << "sim time: " << sim_time << "\n";
        
    exit(EXIT_SUCCESS);
}
