
#include <chrono>

#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <fcntl.h>

#include <queue>
#include <map>
#include <vector>

#include "rf_verilated.h"

#include "VNV_nvdla.h"
#include "rtlflow.h"

#include "../spec/defs/project.h"

int main(int argc, char** argv) {
  const size_t NUM_CYCLES = std::stoi(argv[1]);
  const size_t NUM_TRIALS = std::stoi(argv[2]);

  RF::Verilated::commandArgs(argc, argv);

  std::chrono::time_point<std::chrono::steady_clock> tic;
  std::chrono::time_point<std::chrono::steady_clock> toc;
  float duration{0.0f};
  //RF::VNV_nvdla* dut = new RF::VNV_nvdla;
  RF::VNV_nvdla* dut;
  hipMallocManaged(&dut, sizeof(RF::VNV_nvdla));
  RF::RTLflow rtlflow(dut);
  rtlflow.initialize();

  for (size_t c = 0; c < 100; ++c) {
    rtlflow.run();
  }

  tic = std::chrono::steady_clock::now();
  for(size_t t = 0; t < NUM_TRIALS; ++t) {
    for (size_t c = 0; c < NUM_CYCLES; ++c) {
      rtlflow.run();
      rtlflow.run();
    }
  }
  toc = std::chrono::steady_clock::now();

  duration =  (std::chrono::duration_cast<std::chrono::milliseconds>(toc - tic).count() / NUM_TRIALS) / 1000.0f;
  std::ofstream out("./estimated_result.out", std::ios_base::app);
  out << duration << '\n';
  return 0;
}
