#include "hip/hip_runtime.h"
#include <rf_verilated.h>
#include <rtlflow.h>
#include <rf_heavy.h>
#include <iostream>

#include "mm.h"

using namespace std;

vluint64_t main_time = 0;       // Current simulation time
        // This is a 64-bit integer to reduce wrap over issues and
        // allow modulus.  You can also use a double, if you wish.
RF::VTile* top; 

double sc_time_stamp () { // Called by $time in Verilog
  return main_time;       // converts to double, to match
                          // what SystemC does
}

// TODO Provide command-line options like vcd filename, timeout count, etc.
const long timeout = 100000000L;

void tick(
  const size_t NUM_LINES,
  std::vector<RF::RTLflow>& rtlflows,
  std::vector<std::vector<mm_magic_t*>>& mems
) {

  for(size_t l = 0; l < NUM_LINES; ++l) {
    for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
      *rtlflows[l].get(top->clock, b) = 1;
    }

    rtlflows[l].run();
    main_time++;

    for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
      *rtlflows[l].get(top->io_nasti_aw_ready, b) = mems[l][b]->aw_ready();
      *rtlflows[l].get(top->io_nasti_ar_ready, b) = mems[l][b]->ar_ready();
      *rtlflows[l].get(top->io_nasti_w_ready, b) = mems[l][b]->w_ready();
      *rtlflows[l].get(top->io_nasti_b_valid, b) = mems[l][b]->b_valid();
      *rtlflows[l].get(top->io_nasti_b_bits_id, b) = mems[l][b]->b_id();
      *rtlflows[l].get(top->io_nasti_b_bits_resp, b) = mems[l][b]->b_resp();
      *rtlflows[l].get(top->io_nasti_r_valid, b) = mems[l][b]->r_valid();
      *rtlflows[l].get(top->io_nasti_r_bits_id, b) = mems[l][b]->r_id();
      *rtlflows[l].get(top->io_nasti_r_bits_resp, b) = mems[l][b]->r_resp();
      *rtlflows[l].get(top->io_nasti_r_bits_last, b) = mems[l][b]->r_last();
      memcpy(rtlflows[l].get(top->io_nasti_r_bits_data, b), mems[l][b]->r_data(), 8);

      mems[l][b]->tick(
        *rtlflows[l].get(top->reset, b),
        *rtlflows[l].get(top->io_nasti_ar_valid, b),
        *rtlflows[l].get(top->io_nasti_ar_bits_addr, b),
        *rtlflows[l].get(top->io_nasti_ar_bits_id, b),
        *rtlflows[l].get(top->io_nasti_ar_bits_size, b),
        *rtlflows[l].get(top->io_nasti_ar_bits_len, b),

        *rtlflows[l].get(top->io_nasti_aw_valid, b),
        *rtlflows[l].get(top->io_nasti_aw_bits_addr, b),
        *rtlflows[l].get(top->io_nasti_aw_bits_id, b),
        *rtlflows[l].get(top->io_nasti_aw_bits_size, b),
        *rtlflows[l].get(top->io_nasti_aw_bits_len, b),

        *rtlflows[l].get(top->io_nasti_w_valid, b),
        *rtlflows[l].get(top->io_nasti_w_bits_strb, b),
        rtlflows[l].get(top->io_nasti_w_bits_data, b),
        *rtlflows[l].get(top->io_nasti_w_bits_last, b),

        *rtlflows[l].get(top->io_nasti_r_ready, b),
        *rtlflows[l].get(top->io_nasti_b_ready, b)
      );
      
      *rtlflows[l].get(top->clock, b) = 0;
    }
    rtlflows[l].run();
  }
//#if VM_TRACE
  //if (tfp) tfp->dump((double) main_time);
//#endif // VM_TRACE
  main_time++;
}

int main(int argc, char** argv) {
  RF::Verilated::commandArgs(argc, argv);   // Remember args

  const size_t NUM_TESTBENCHES = std::stoi(argv[1]);
  const size_t NUM_CYCLES      = std::stoi(argv[2]);
  const size_t NUM_LINES       = NUM_TESTBENCHES / RF::BATCH_SIZE;
  std::string dir{argv[3]}; 

  top = new RF::VTile; // target design
  std::vector<std::vector<mm_magic_t*>> mems(NUM_LINES);
  std::vector<RF::RTLflow> rtlflows(NUM_LINES, top);

  for(size_t l = 0; l < NUM_LINES; ++l) {
    mems[l].resize(RF::BATCH_SIZE);
    for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
      mems[l][b] = new mm_magic_t(1L << 32, 8); // target memory
      load_mem(mems[l][b]->get_data(), std::string(dir+"tb"+std::to_string(l * RF::BATCH_SIZE + b)+".hex").c_str()); // load hex

    }
    rtlflows[l].initialize();
  }


//#if VM_TRACE			// If verilator was invoked with --trace
  //Verilated::traceEverOn(true);	// Verilator must compute traced signals
  //VL_PRINTF("Enabling waves...\n");
  //tfp = new VerilatedVcdC;
  //top->trace(tfp, 99);	// Trace 99 levels of hierarchy
  //tfp->open(argc > 2 ? argv[2] : "dump.vcd"); // Open the dump file
//#endif

  cout << "Starting simulation!\n";

  // reset
  for(size_t l = 0; l < NUM_LINES; ++l) {
    for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
      *rtlflows[l].get(top->reset, b) = 1;
    }
  }
  
  for (size_t i = 0; i < 5 ; i++) {
    tick(NUM_LINES, rtlflows, mems);
  }

  // start
  for(size_t l = 0; l < NUM_LINES; ++l) {
    for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
      *rtlflows[l].get(top->reset, b) = 0;
      *rtlflows[l].get(top->io_host_fromhost_bits, b) = 0;
      *rtlflows[l].get(top->io_host_fromhost_valid, b) = 0;
    }
  }

  //for(size_t c = 0; c < NUM_CYCLES; ++c) {
    //tick(NUM_LINES, rtlflows, mems);
  //}
  do {
    //tick();
    tick(NUM_LINES, rtlflows, mems);
  } while(!(*rtlflows[0].get(top->io_host_tohost, 0)) && main_time < timeout);


  int retcode = *rtlflows[0].get(top->io_host_tohost, 0) >> 1;

  // Run for 10 more clocks
  for (size_t i = 0 ; i < 10 ; i++) {
    tick(NUM_LINES, rtlflows, mems);
  }

  if (main_time >= timeout) {
    cerr << "Simulation terminated by timeout at time " << main_time
         << " (cycle " << main_time / 10 << ")"<< endl;
    return EXIT_FAILURE;
  } else {
    cerr << "Simulation completed at time " << main_time <<
           " (cycle " << main_time / 10 << ")"<< endl;
    if (retcode) {
      cerr << "TOHOST = " << retcode << endl;
    }
  }

//#if VM_TRACE
  //if (tfp) tfp->close();
  //delete tfp;
//#endif
  //delete top;
  //delete mem;

  cout << "Finishing simulation!\n";

  return retcode == 0 ? EXIT_SUCCESS : EXIT_FAILURE;
  //return 0;
}

