#include "hip/hip_runtime.h"
#include <rf_verilated.h>
#include <rtlflow.h>
#include <rf_heavy.h>
#include <iostream>

#include "mm.h"

using namespace std;
RF::VTile* top; 

const long timeout = 100000000L;

void build_task_graph(
  taro::TaroCBV4& taro,
  const size_t NUM_LINES,
  const size_t NUM_PIPES,
  std::vector<RF::RTLflow>& rtlflows,
  std::vector<std::vector<mm_magic_t*>>& mems
) {


  for(size_t l = 0; l < NUM_LINES; ++l) {
    // reset
    auto reset_t = taro.emplace([&, l]() {
      for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
        *rtlflows[l].get(top->reset, b) = 1;
      }
    });
    // 5 ticks
    std::vector<std::pair<taro::TaskHandle, taro::TaskHandle>> ticks(5);
    for (size_t t = 0; t < 5 ; t++) {

      auto t1 = taro.emplace([&, l]() {
        for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
          *rtlflows[l].get(top->clock, b) = 1;
        }
      });

      auto t2 = taro.emplace([&, l]() {
        for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
          *rtlflows[l].get(top->io_nasti_aw_ready, b) = mems[l][b]->aw_ready();
          *rtlflows[l].get(top->io_nasti_ar_ready, b) = mems[l][b]->ar_ready();
          *rtlflows[l].get(top->io_nasti_w_ready, b) = mems[l][b]->w_ready();
          *rtlflows[l].get(top->io_nasti_b_valid, b) = mems[l][b]->b_valid();
          *rtlflows[l].get(top->io_nasti_b_bits_id, b) = mems[l][b]->b_id();
          *rtlflows[l].get(top->io_nasti_b_bits_resp, b) = mems[l][b]->b_resp();
          *rtlflows[l].get(top->io_nasti_r_valid, b) = mems[l][b]->r_valid();
          *rtlflows[l].get(top->io_nasti_r_bits_id, b) = mems[l][b]->r_id();
          *rtlflows[l].get(top->io_nasti_r_bits_resp, b) = mems[l][b]->r_resp();
          *rtlflows[l].get(top->io_nasti_r_bits_last, b) = mems[l][b]->r_last();
          memcpy(rtlflows[l].get(top->io_nasti_r_bits_data, b), mems[l][b]->r_data(), 8);

          mems[l][b]->tick(
            *rtlflows[l].get(top->reset, b),
            *rtlflows[l].get(top->io_nasti_ar_valid, b),
            *rtlflows[l].get(top->io_nasti_ar_bits_addr, b),
            *rtlflows[l].get(top->io_nasti_ar_bits_id, b),
            *rtlflows[l].get(top->io_nasti_ar_bits_size, b),
            *rtlflows[l].get(top->io_nasti_ar_bits_len, b),

            *rtlflows[l].get(top->io_nasti_aw_valid, b),
            *rtlflows[l].get(top->io_nasti_aw_bits_addr, b),
            *rtlflows[l].get(top->io_nasti_aw_bits_id, b),
            *rtlflows[l].get(top->io_nasti_aw_bits_size, b),
            *rtlflows[l].get(top->io_nasti_aw_bits_len, b),

            *rtlflows[l].get(top->io_nasti_w_valid, b),
            *rtlflows[l].get(top->io_nasti_w_bits_strb, b),
            rtlflows[l].get(top->io_nasti_w_bits_data, b),
            *rtlflows[l].get(top->io_nasti_w_bits_last, b),

            *rtlflows[l].get(top->io_nasti_r_ready, b),
            *rtlflows[l].get(top->io_nasti_b_ready, b)
          );
          
          *rtlflows[l].get(top->clock, b) = 0;
        }
      });
      auto sim1 = rtlflows[l].create_sim_t();
      auto sim2 = rtlflows[l].create_sim_t();
      t1.precede(sim1);
      sim1.precede(t2);
      t2.precede(sim2);
      ticks[t].first = t1;
      ticks[t].second = sim2;
    }

    for(size_t t = 0; t < 4; ++t) {
      ticks[t].second.precede(ticks[t + 1].first);
    }

    // start
    auto start_t = taro.emplace([&, l]() mutable {
      for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
        *rtlflows[l].get(top->reset, b) = 0;
        *rtlflows[l].get(top->io_host_fromhost_bits, b) = 0;
        *rtlflows[l].get(top->io_host_fromhost_valid, b) = 0;
      }
    });

    std::vector<std::pair<taro::TaskHandle, taro::TaskHandle>> pipes(NUM_PIPES);
    for(size_t p = 0; p < NUM_PIPES; ++p) {
      auto t1 = taro.emplace([&, l]() mutable {
        for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
          *rtlflows[l].get(top->clock, b) = 1;
        }
      });

      auto t2 = taro.emplace([&, l]() {
        for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
          *rtlflows[l].get(top->io_nasti_aw_ready, b) = mems[l][b]->aw_ready();
          *rtlflows[l].get(top->io_nasti_ar_ready, b) = mems[l][b]->ar_ready();
          *rtlflows[l].get(top->io_nasti_w_ready, b) = mems[l][b]->w_ready();
          *rtlflows[l].get(top->io_nasti_b_valid, b) = mems[l][b]->b_valid();
          *rtlflows[l].get(top->io_nasti_b_bits_id, b) = mems[l][b]->b_id();
          *rtlflows[l].get(top->io_nasti_b_bits_resp, b) = mems[l][b]->b_resp();
          *rtlflows[l].get(top->io_nasti_r_valid, b) = mems[l][b]->r_valid();
          *rtlflows[l].get(top->io_nasti_r_bits_id, b) = mems[l][b]->r_id();
          *rtlflows[l].get(top->io_nasti_r_bits_resp, b) = mems[l][b]->r_resp();
          *rtlflows[l].get(top->io_nasti_r_bits_last, b) = mems[l][b]->r_last();
          memcpy(rtlflows[l].get(top->io_nasti_r_bits_data, b), mems[l][b]->r_data(), 8);

          mems[l][b]->tick(
            *rtlflows[l].get(top->reset, b),
            *rtlflows[l].get(top->io_nasti_ar_valid, b),
            *rtlflows[l].get(top->io_nasti_ar_bits_addr, b),
            *rtlflows[l].get(top->io_nasti_ar_bits_id, b),
            *rtlflows[l].get(top->io_nasti_ar_bits_size, b),
            *rtlflows[l].get(top->io_nasti_ar_bits_len, b),

            *rtlflows[l].get(top->io_nasti_aw_valid, b),
            *rtlflows[l].get(top->io_nasti_aw_bits_addr, b),
            *rtlflows[l].get(top->io_nasti_aw_bits_id, b),
            *rtlflows[l].get(top->io_nasti_aw_bits_size, b),
            *rtlflows[l].get(top->io_nasti_aw_bits_len, b),

            *rtlflows[l].get(top->io_nasti_w_valid, b),
            *rtlflows[l].get(top->io_nasti_w_bits_strb, b),
            rtlflows[l].get(top->io_nasti_w_bits_data, b),
            *rtlflows[l].get(top->io_nasti_w_bits_last, b),

            *rtlflows[l].get(top->io_nasti_r_ready, b),
            *rtlflows[l].get(top->io_nasti_b_ready, b)
          );
          
          *rtlflows[l].get(top->clock, b) = 0;
        }
      });

      auto sim1 = rtlflows[l].create_sim_t();
      auto sim2 = rtlflows[l].create_sim_t();
      t1.precede(sim1);
      sim1.precede(t2);
      t2.precede(sim2);

      pipes[p].first = t1;
      pipes[p].second = sim2;
    }

    for(size_t p = 0; p < NUM_PIPES - 1; ++p) {
      pipes[p].second.precede(pipes[p + 1].first);
    }

    reset_t.precede(ticks[0].first);
    ticks[4].second.precede(start_t);
    start_t.precede(pipes[0].first);
  }

}


int main(int argc, char** argv) {
  RF::Verilated::commandArgs(argc, argv);   // Remember args

  const size_t NUM_TESTBENCHES = std::stoi(argv[1]);
  const size_t NUM_CYCLES      = std::stoi(argv[2]);
  const size_t NUM_PIPES       = NUM_CYCLES;
  const size_t NUM_LINES       = NUM_TESTBENCHES / RF::BATCH_SIZE;
  std::string dir{argv[3]}; 
  std::cerr << "start1111\n";

  taro::TaroCBV4 taro{4, 4};
  top = new RF::VTile; // target design
  std::cerr << "start2222\n";
  std::cerr << "num of lines: " << NUM_LINES << "\n";
  std::vector<std::vector<mm_magic_t*>> mems(NUM_LINES, std::vector<mm_magic_t*>(RF::BATCH_SIZE));
  std::cerr << "3333\n";

  std::vector<RF::RTLflow> rtlflows(NUM_LINES, {top, taro});
  for(size_t l = 0; l < NUM_LINES; ++l) {
    for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
      mems[l][b] = new mm_magic_t(1L << 28, 8); // target memory
      load_mem(mems[l][b]->get_data(), std::string(dir+"tb"+std::to_string(l * RF::BATCH_SIZE + b)+".hex").c_str()); // load hex

    }
  }

  cout << "11111!\n";
  for(size_t l = 0; l < NUM_LINES; ++l) {
    rtlflows[l].initialize();
  }
  build_task_graph(taro, NUM_LINES, NUM_PIPES, rtlflows, mems);

  //// reset
  //for(size_t l = 0; l < NUM_LINES; ++l) {
    //for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
      //*rtlflows[l].get(top->reset, b) = 1;
    //}
  //}
  
  //for (size_t i = 0; i < 5 ; i++) {
    //tick(NUM_LINES, rtlflows, mems);
  //}


  cout << "2222!\n";

  cout << "Starting simulation!\n";

  taro.schedule();
  taro.wait();

  delete top;
  for(size_t l = 0; l < NUM_LINES; ++l) {
    for(size_t b = 0; b < RF::BATCH_SIZE; ++b) {
      delete mems[l][b];
    }
  }

  cout << "Finishing simulation!\n";

  //return retcode == 0 ? EXIT_SUCCESS : EXIT_FAILURE;
  return 0;
}

